#include "hip/hip_runtime.h"
#include "moments.h"
#include "mycudaheaders.h"
#include <assert.h>
#include <chrono>


using namespace std::chrono;

inline
hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

//Y moments
//X data
__global__ void
optimizedKernel(int *const moments, const int *const data, const unsigned numMoments, const unsigned numData) {

    __shared__ int localmoments[FACTOR][TILE_SIZE][TILE_SIZE];
    __shared__ int localdata[FACTOR][TILE_SIZE][TILE_SIZE];

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;


    for (size_t f = 0; f < FACTOR; f++) {
        if ((f * TILE_SIZE + x) * TILE_SIZE + threadIdx.y < numData * REC_SIZE_WORDS) {
            localdata[f][threadIdx.x][threadIdx.y] = data[(f * TILE_SIZE + x) * TILE_SIZE + threadIdx.y];
        } else {
            localdata[f][threadIdx.x][threadIdx.y] = 0;
        }
        if ((f * TILE_SIZE + y) * TILE_SIZE + threadIdx.x < numMoments * REC_SIZE_WORDS) {
            localmoments[f][threadIdx.y][threadIdx.x] = moments[(f * TILE_SIZE + y) * TILE_SIZE + threadIdx.x];
        } else {
            localmoments[f][threadIdx.y][threadIdx.x] = 0;
        }
    }
    __syncthreads();

    for (unsigned yf = 0; yf < FACTOR; yf++) {
        for (unsigned yz = 0; yz < RECS_IN_TILE; yz++) {
            T acc = 0;
            for (unsigned xf = 0; xf < FACTOR; xf++) {
                for (unsigned xz = 0; xz < RECS_IN_TILE; xz++) {
                    bool andcond = ((xf * TILE_SIZE + x) * TILE_SIZE + xz * REC_SIZE_WORDS + KEY_SIZE_WORDS <
                                    numData * REC_SIZE_WORDS) &&
                                   ((yf * TILE_SIZE + y) * TILE_SIZE + yz * REC_SIZE_WORDS + KEY_SIZE_WORDS <
                                    numMoments * REC_SIZE_WORDS);
                    for (unsigned k = 0; andcond && k < KEY_SIZE_WORDS; k++) {
                        andcond =
                                andcond &&
                                ((localmoments[yf][threadIdx.y][yz * REC_SIZE_WORDS + k] &
                                  localdata[xf][threadIdx.x][xz * REC_SIZE_WORDS + k]) ==
                                 localmoments[yf][threadIdx.y][yz * REC_SIZE_WORDS + k]);
                    }
                    if (andcond)
                        acc += *(T *) (&localdata[xf][threadIdx.x][xz * REC_SIZE_WORDS + KEY_SIZE_WORDS]);
                }
            }
            if (acc != 0) {
                atomicAdd(reinterpret_cast<T *>(moments + (yf * TILE_SIZE + y) * TILE_SIZE + yz * REC_SIZE_WORDS +
                                                KEY_SIZE_WORDS), acc);
            }
        }
    }
}

__global__ void
computeKernel(int *moments, const int *data, const unsigned numMoments, const unsigned jstart,
              const unsigned jend) {
    __shared__ int localmoments[ENTRIES_IN_BLOCK][REC_SIZE_WORDS + 1];
    __shared__ int localdata[ENTRIES_IN_BLOCK][REC_SIZE_WORDS + 1];

    const unsigned blockMomentStart = blockIdx.x * ENTRIES_IN_BLOCK;
    const unsigned yid = threadIdx.x >> 4;
    const unsigned xid = threadIdx.x & 15;

    for (unsigned int i = yid;
         i < ENTRIES_IN_BLOCK && (blockMomentStart + i) < numMoments; i += 8) {
        localmoments[i][xid] = moments[(blockMomentStart + i) * REC_SIZE_WORDS + xid];

    }
    __syncthreads();
    for (unsigned joffset = jstart; joffset < jend; joffset += ENTRIES_IN_BLOCK) {
        //Copy DATA into shared memory
        for (unsigned int j = yid;
             j < ENTRIES_IN_BLOCK && (joffset + j) < jend; j += 8) {
            localdata[j][xid] = data[(joffset + j) * REC_SIZE_WORDS + xid];
        }
        __syncthreads();
        for (unsigned int i = threadIdx.x;
             i < ENTRIES_IN_BLOCK && (blockMomentStart + i) < numMoments; i += CORES_PER_BLOCK) {
            for (unsigned int j = 0; j < ENTRIES_IN_BLOCK && (joffset + j) < jend; j++) {
                unsigned int k = 0;
                for (; k < KEY_SIZE_WORDS; k++) {
                    if ((localmoments[i][k] & localdata[j][k]) != localmoments[i][k]) {
                        break;
                    }
                }
                if (k == KEY_SIZE_WORDS) {
                    *(T *) (localmoments[i] + KEY_SIZE_WORDS) += *(T *) (localdata[j] + KEY_SIZE_WORDS);
                }
            }
        }
        __syncthreads();
    }
    for (unsigned int i = threadIdx.x;
         i < ENTRIES_IN_BLOCK && (blockMomentStart + i) < numMoments; i += CORES_PER_BLOCK) {
        *(T *) (moments + (blockMomentStart + i) * REC_SIZE_WORDS + KEY_SIZE_WORDS) = *(T *) (
                localmoments[i] +
                KEY_SIZE_WORDS);
    }

}

void computeCUDA(TypedCuboid &moments, const TypedCuboid &data) {
    const unsigned int numBlocks = moments.numRows / ENTRIES_IN_BLOCK + 1;
    const int halfmemorywords = (6UL * 1000 * 1000 * 1000) / sizeof(int);
    const int halfmemoryrecords = halfmemorywords / REC_SIZE_WORDS;

    int numDevices;
    hipGetDeviceCount(&numDevices);
    numDevices = 1;
    printf("Number of GPU detected = %d\n", numDevices);
    int **gpu_moments = new int *[numDevices];
    int **gpu_data = new int *[numDevices];
    auto startTime = high_resolution_clock::now();
    //Allocate memory on all devices
    for (int devId = 0; devId < numDevices; devId++) {
        checkCuda(hipSetDevice(devId));
        checkCuda(hipMalloc(gpu_moments + devId, halfmemoryrecords * REC_SIZE_WORDS * sizeof(int)));
        checkCuda(hipMalloc(gpu_data + devId, halfmemoryrecords * REC_SIZE_WORDS * sizeof(int)));
    }


    size_t numMomentBlocks = (1 + moments.numRows / (numDevices * halfmemoryrecords));
    size_t numDataBlocks = (1 + data.numRows / halfmemoryrecords);
    size_t totalSteps = numDataBlocks * numMomentBlocks;
    size_t stepCount = 0;
    printf("Total steps = %lu\n", totalSteps);
    printf("Kernel blocks = %lu BlockSize = %lu\n", numBlocks, CORES_PER_BLOCK);
    //For each block of moments to be computed
    for (size_t momentoffset = 0;
         momentoffset < moments.numRows; momentoffset += numDevices * halfmemoryrecords) {
        int curMaxDevId = numDevices;
        //Copy the block of moments to be computed to respective devices
        for (int devId = 0; devId < numDevices; devId++) {
            size_t thisdeviceMomentsOffset = momentoffset + halfmemoryrecords * devId;
            if (thisdeviceMomentsOffset >= moments.numRows) {
                curMaxDevId = devId;
                break;
            }
            checkCuda(hipSetDevice(devId));
            size_t numMomentsRemaining = std::min<size_t>(halfmemoryrecords, moments.numRows - thisdeviceMomentsOffset);
            printf("NumMomentsRemaining for device %d = %lu\n", devId, numMomentsRemaining);
            if (numMomentsRemaining > 0)
                checkCuda(hipMemcpy(gpu_moments[devId], moments.ptr + thisdeviceMomentsOffset * REC_SIZE_WORDS,
                                     numMomentsRemaining * REC_SIZE_WORDS * sizeof(int),
                                     hipMemcpyHostToDevice));
        }

        //For each block of data
        for (size_t dataoffset = 0; dataoffset < data.numRows; dataoffset += halfmemoryrecords) {

            //Copy the block of data to be computed to the devices. Same for all devices
            size_t numDataRemaining = std::min<size_t>(halfmemoryrecords, data.numRows - dataoffset);
            printf("NumDataRemaining = %lu \n", numDataRemaining);
            size_t numDataPerCent = numDataRemaining / 100 + 1;
            for (int devId = 0; devId < curMaxDevId; devId++) {
                checkCuda(hipSetDevice(devId));
                checkCuda(hipMemcpy(gpu_data[devId], data.ptr + dataoffset * REC_SIZE_WORDS,
                                     numDataRemaining * REC_SIZE_WORDS * sizeof(int),
                                     hipMemcpyHostToDevice));
            }
            //Launch smaller kernels to track progress
            for (size_t start = 0; start < numDataRemaining; start += numDataPerCent) {
                for (int devId = 0; devId < curMaxDevId; devId++) {
                    checkCuda(hipSetDevice(devId));
                    size_t thisdeviceMomentsOffset = momentoffset + halfmemoryrecords * devId;
                    size_t numMomentsRemaining = std::min<size_t>(halfmemoryrecords,
                                                                  moments.numRows - thisdeviceMomentsOffset);

                    computeKernel<<<numBlocks, CORES_PER_BLOCK>>>(gpu_moments[devId], gpu_data[devId], numMomentsRemaining, start, std::min(
                            start + numDataPerCent, numDataRemaining));
                }


//            for (int devId = 0; devId < curMaxDevId; devId++) {
//                size_t thisdeviceMomentsOffset = momentoffset + halfmemoryrecords * devId;
//                size_t numMomentsRemaining = std::min<size_t>(halfmemoryrecords,
//                                                              moments.numRows - thisdeviceMomentsOffset);
//                checkCuda(hipSetDevice(devId));
//                dim3 threads_per_block(TILE_SIZE, TILE_SIZE);
//                dim3 blocks_per_grid(1, 1);
//                blocks_per_grid.y = std::ceil(static_cast<double>(numMomentsRemaining) /
//                                              static_cast<double>(TILE_SIZE * RECS_IN_TILE * FACTOR));
//                blocks_per_grid.x = std::ceil(static_cast<double>(numDataRemaining) /
//                                              static_cast<double>(TILE_SIZE * RECS_IN_TILE * FACTOR));
//                printf("BLOCKS = %d %d \n", blocks_per_grid.x, blocks_per_grid.y);
//                optimizedKernel<<<blocks_per_grid, threads_per_block>>>(gpu_moments[devId], gpu_data[devId], numMomentsRemaining, numDataRemaining);
//            }
                for (int devId = 0; devId < curMaxDevId; devId++) {
                    checkCuda(hipSetDevice(devId));
                    checkCuda(hipDeviceSynchronize());
                    stepCount++;
                    printf("Step %lu / (%lu x %lu x 100) \n", stepCount, numMomentBlocks, numDataBlocks);
                }
            }
        }

        //Copy the result for the block of moments from respective devices
        for (int devId = 0; devId < curMaxDevId; devId++) {
            size_t thisdeviceMomentsOffset = momentoffset + halfmemoryrecords * devId;
            checkCuda(hipSetDevice(devId));
            size_t numMomentsRemaining = std::min<size_t>(halfmemoryrecords, moments.numRows - thisdeviceMomentsOffset);
            checkCuda(hipMemcpy(moments.ptr + thisdeviceMomentsOffset * REC_SIZE_WORDS, gpu_moments[devId],
                                 numMomentsRemaining * REC_SIZE_WORDS * sizeof(int),
                                 hipMemcpyDeviceToHost));
        }
    }

    auto endTime = high_resolution_clock::now();
    auto duration = duration_cast<seconds>(endTime - startTime).count();
    printf("Computation on GPU took %lu seconds \n", duration);

    for (int devId = 0; devId < numDevices; devId++) {
        checkCuda(hipSetDevice(devId));
        checkCuda(hipFree(gpu_moments[devId]));
        checkCuda(hipFree(gpu_data[devId]));
    }
    delete[] gpu_moments;
    delete[] gpu_data;
}
